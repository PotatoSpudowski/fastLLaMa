#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <atomic>
#include "ggml-cuda.h"

typedef uint16_t ggml_fp16_t;
static_assert(sizeof(__half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define QK4_0 32
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK4_2 16
typedef struct {
    __half  d;              // delta
    uint8_t qs[QK4_2 / 2];  // nibbles / quants
} block_q4_2;
static_assert(sizeof(block_q4_2) == sizeof(ggml_fp16_t) + QK4_2 / 2, "wrong q4_2 block size/padding");

#define QK4_3 16
typedef struct {
    __half  d;              // delta
    __half  m;              // min
    uint8_t qs[QK4_3 / 2];  // nibbles / quants
} block_q4_3;
static_assert(sizeof(block_q4_3) == 2 * sizeof(ggml_fp16_t) + QK4_3 / 2, "wrong q4_3 block size/padding");

static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_0 + l + 0] = v0;
        y[i*QK4_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_1 + l + 0] = v0;
        y[i*QK4_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_2(const void * vx, float * y) {
    const block_q4_2 * x = (const block_q4_2 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_2; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_2 + l + 0] = v0;
        y[i*QK4_2 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_3(const void * vx, float * y) {
    const block_q4_3 * x = (const block_q4_3 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_3; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_3 + l + 0] = v0;
        y[i*QK4_3 + l + 1] = v1;
    }
}

void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_0;
    dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_1;
    dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q4_2_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_2;
    dequantize_block_q4_2<<<nb, 1, 0, stream>>>(vx, y);
}

void dequantize_row_q4_3_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_3;
    dequantize_block_q4_3<<<nb, 1, 0, stream>>>(vx, y);
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 16

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

hipblasHandle_t g_cublasH = NULL;
hipStream_t g_cudaStream = NULL;

void ggml_init_cublas(void) {
    if (g_cublasH == NULL) {
        // create cublas handle, bind a stream
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));

        CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStream, hipStreamNonBlocking));

        CUBLAS_CHECK(hipblasSetStream(g_cublasH, g_cudaStream));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, NULL));
    }
}